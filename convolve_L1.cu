#include "hip/hip_runtime.h"
#include "math.h"
#include "stdlib.h"
#include "stdio.h"
#include "time.h"
#include "sys/time.h"
#include "hip/hip_runtime.h"
#include "image_template.h"

__global__
void convolve_hor(float *image, int width, int height, float *mask, int mask_width, float* out_image){
	
	int i, j, k;
	int sharedwidth = blockDim.x + 2 * floorf(mask_width/2);
	float sum=0;
	extern __shared__ float AShared[];

	i=blockIdx.x*blockDim.x + threadIdx.x;
	j=blockIdx.y*blockDim.y + threadIdx.y;

	// Collab load
	AShared[threadIdx.x * sharedwidth + (threadIdx.y + mask_width/2)] = image[i*width+j]; 
	__syncthreads();
		

	// Load Left aprons
	if(threadIdx.y >= blockDim.y - (mask_width/2)){
		if(blockIdx.y >= 1){
			AShared[threadIdx.x*sharedwidth + threadIdx.y - blockDim.y + (mask_width/2)]
				= image[i*width + (blockIdx.y-1) * blockDim.y + threadIdx.y];
		}
		else{
			AShared[threadIdx.x * sharedwidth + threadIdx.y - blockDim.y + (mask_width/2)] = 0;
		}
	}
	__syncthreads();

	// Load right aprons
	if(threadIdx.y < (mask_width/2)){
		if(blockIdx.y < gridDim.y-1){
			AShared[threadIdx.x*sharedwidth + mask_width/2 + blockDim.y + threadIdx.y] 
				= image[i * width + (blockIdx.y + 1)* blockDim.y + threadIdx.y];
		}
		else{
			AShared[threadIdx.x * sharedwidth + mask_width/2 + blockDim.y + threadIdx.y] = 0;
		}
	}		
	__syncthreads();

	// Convolve
	for(k=0; k<mask_width; k++){
		sum += AShared[threadIdx.x * sharedwidth + (threadIdx.y+k)]*mask[k];
	}

	// Write results
	out_image[i*width+j] = sum;
	__syncthreads();
}

__global__
void convolve_ver(float *image, int width, int height, float *mask, int mask_width, float* out_image){
	int i, j, k;
	int sharedwidth = blockDim.x;
	float sum=0;
	extern __shared__ float AShared[];

	i=blockIdx.x*blockDim.x + threadIdx.x;
	j=blockIdx.y*blockDim.y + threadIdx.y;

	// Collab load
	AShared[(threadIdx.x+mask_width/2) * sharedwidth + (threadIdx.y)] = image[i*width+j]; 
	__syncthreads();
		

	// Load top aprons
	if(threadIdx.x >= blockDim.x - (mask_width/2)){
		if(blockIdx.x >= 1){
			AShared[(threadIdx.x-blockDim.x+(mask_width/2))*sharedwidth + threadIdx.y]
				= image[((blockIdx.x-1)*blockDim.x+threadIdx.x)*width + j];
		}
		else{
			AShared[(threadIdx.x-blockDim.x+(mask_width/2))*sharedwidth + threadIdx.y]=0;
		}
	}
	__syncthreads();

	// Load bot aprons
	if(threadIdx.x < (mask_width/2)){
		if(blockIdx.x < gridDim.x-1){
			AShared[(threadIdx.x+blockDim.x +mask_width/2)*sharedwidth +threadIdx.y] 
				= image[((blockIdx.x+1)*blockDim.x+threadIdx.x)*width + j];
		}
		else{
			AShared[(threadIdx.x+blockDim.x + mask_width/2)*sharedwidth +threadIdx.y]=0; 
		}
	}		
	__syncthreads();

	// Convolve
	for(k=0; k<mask_width; k++){
		sum += AShared[(threadIdx.x+k) * sharedwidth + (threadIdx.y)]*mask[k];
	}

	// Write results
	out_image[i*width+j] = sum;
	__syncthreads();

}

void create_gaussians(float **g_kernel, float **dg_kernel, float sigma, int *w){
	float a = ceil(2.5*sigma-0.5);
	int sum = 0;
	
	*w=2*a+1;
	*g_kernel=(float*)malloc(sizeof(float)*(*w));

	// Calculate gaussian	
	for(int i=0; i<(*w); i++){
		(*g_kernel)[i] = exp((-1*(i-a)*(i-a))/
			  (2*sigma*sigma));
		sum+=(*g_kernel)[i];			   
	}
	
	// Normalize
	for(int i=0; i<(*w); i++){
		(*g_kernel)[i]/=sum;	
	}

	// Calculate Derivative
	sum = 0;
	
	*dg_kernel=(float*)malloc(sizeof(float)*(*w));
	
	for(int i=0; i<(*w); i++){
		(*dg_kernel)[i] = (-1*(i-a))*exp((-1*(i-a)*(i-a))/
			  (2*sigma*sigma));
		sum-=i*(*dg_kernel)[i];			   
	}
	
	// Normalize
	for(int i=0; i<(*w); i++){
		(*dg_kernel)[i]/=sum;	
	}
	
}

void print_matrix(float *matrix, int height, int width){
	for(int i=0; i<height; i++){
		for(int j=0; j<width; j++){
			printf("%.3f ", *(matrix+(i*width)+j));
		}
		printf("\n");
	}
}

int main(int argc, char **argv){
	if(argc != 3)
		printf("convolution <file> <sigma>\n");
	else{
		int height, width, k_width;
		struct timeval start, end;
		
		// CPU buffer for orig_img
		float *org_img;

		// GPU device buffer for original img
		float *d_org_img;
	
		//CPU host buffers for the final output
		float *vertical_gradient, *horizontal_gradient, *temp_gradient, *temp_hor_gradient;

		//GPU host buffers for the final output
		float *d_vertical_gradient, *d_horizontal_gradient;
		
		// GPU buffers to hold intermediate convolution results
		float *d_temp_horizontal, *d_temp_vertical;

		// CPU host buffers to store convolution masks
		float *gaussian_kernel, *gaussian_deriv;

		// GPU device buffers to store the convolutions masks
		float *d_gaussian_kernel, *d_gaussian_deriv;
		
		read_image_template(argv[1],
				    &org_img,
				    &width,
				    &height);
		
		create_gaussians(&gaussian_kernel, &gaussian_deriv, atof(argv[2]), &k_width);
	
		printf("Gaussian Kernel:\n");
		print_matrix(gaussian_kernel, 1, k_width);
		printf("Derivative Kernel:\n");
		print_matrix(gaussian_deriv,1,k_width);

		// CPU host mallocs for GPU buffers
		hipMalloc((void**)&d_org_img, sizeof(float)*width*height);
		hipMalloc((void**)&d_temp_horizontal, sizeof(float)*width*height);
		hipMalloc((void**)&d_temp_vertical, sizeof(float)*width*height);
		hipMalloc((void**)&d_horizontal_gradient, sizeof(float)*width*height);
		hipMalloc((void**)&d_vertical_gradient, sizeof(float)*width*height);
		hipMalloc((void**)&d_gaussian_kernel, sizeof(float)*k_width);
		hipMalloc((void**)&d_gaussian_deriv, sizeof(float)*k_width);

		gettimeofday(&start, NULL);
		// Offload all of the data to GPU device for convolution
		hipMemcpy(d_org_img, org_img, sizeof(float)*width*height, hipMemcpyHostToDevice);
		hipMemcpy(d_gaussian_kernel, gaussian_kernel, sizeof(float)*k_width, hipMemcpyHostToDevice);
		hipMemcpy(d_gaussian_deriv, gaussian_deriv, sizeof(float)*k_width, hipMemcpyHostToDevice);

		int block_dim = 16;
		dim3 dmGrid(ceil(height/block_dim), ceil(width/block_dim), 1);
		dim3 dmBlock(block_dim, block_dim, 1);

		// Vertical Gradient
		convolve_hor<<<dmGrid,dmBlock, sizeof(float)*16*(16+2*floor(k_width/2))>>>
				(d_org_img, width, height, d_gaussian_kernel, k_width, d_temp_vertical);	
		convolve_ver<<<dmGrid,dmBlock,sizeof(float)*16*(16+2*floor(k_width/2))>>>
				(d_temp_vertical, width, height, d_gaussian_deriv, k_width, d_vertical_gradient);	

		// Horizontal Gradient
		convolve_ver<<<dmGrid,dmBlock,sizeof(float)*16*(16+2*floor(k_width/2))>>>
				(d_org_img, width, height, d_gaussian_kernel, k_width, d_temp_horizontal);	
		convolve_hor<<<dmGrid,dmBlock,sizeof(float)*16*(16+2*floor(k_width/2))>>>
				(d_temp_horizontal, width, height, d_gaussian_deriv, k_width, d_horizontal_gradient);	
		
		horizontal_gradient = (float*)malloc(sizeof(float)*height*width);
		vertical_gradient = (float*)malloc(sizeof(float)*height*width);
		temp_gradient = (float*)malloc(sizeof(float)*height*width);	
		temp_hor_gradient = (float*)malloc(sizeof(float)*height*width);	

		hipMemcpy(horizontal_gradient, d_horizontal_gradient, sizeof(float)*width*height, hipMemcpyDeviceToHost);
		hipMemcpy(vertical_gradient, d_vertical_gradient, sizeof(float)*width*height, hipMemcpyDeviceToHost);
		
		gettimeofday(&end, NULL);

		write_image_template("h_gradient_L1.pgm", horizontal_gradient, width, height);
		write_image_template("v_gradient_L1.pgm", vertical_gradient, width, height);

		printf("%ld\n", (end.tv_sec *1000000 + end.tv_usec)-(start.tv_sec * 1000000 + start.tv_usec));
	
		// Cuda Free
		hipFree(d_org_img);
		hipFree(d_temp_horizontal);
		hipFree(d_temp_vertical);
		hipFree(d_horizontal_gradient);
		hipFree(d_vertical_gradient);
		hipFree(d_gaussian_kernel);
		hipFree(d_gaussian_deriv);
	}
}
